

#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void random_uniform_int_kernel(const int min, const int max, int* x, const size_t tb, const size_t N) {
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * tb;
    if (idx >= N)
        return;

    hiprandState rnd_state;
    hiprand_init(clock64(), idx, 0, &rnd_state);
    for (size_t i = idx; i < idx + tb; ++i) {
        x[i] = hiprand_uniform(&rnd_state) * (max - min) + min;
    }
}

__global__ void random_uniform_float_kernel(const float min, const float max, float* x, const size_t tb, const size_t N) {
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * tb;
    if (idx >= N) {
        return;
    }

    hiprandState rnd_state;
    hiprand_init(clock64(), idx, 0, &rnd_state);
    for (int i = idx; i < idx + tb; ++i) {
        x[i] = hiprand_uniform(&rnd_state) * (max - min) + min;
    }
}

void random_uniform(const int min, const int max, int* x, const size_t N) {
    const size_t block_size = 512;
    const size_t tb = 128;
    const size_t grid_size = N / (block_size * tb) + 1;
    random_uniform_int_kernel<<<grid_size, block_size>>>(min, max, x, tb, N);
}

void random_uniform(const float min, const float max, float* x, const size_t N) {
    const size_t block_size = 1024;
    const size_t tb = 128;
    const size_t grid_size = N / (block_size * tb) + 1;
    random_uniform_float_kernel<<<grid_size, block_size>>>(min, max, x, tb, N);
}

